
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

/*
// created by: Isayah Reed
// Before compiling this program, set the cuda library path. Example:
// $> export LD_LIBRARY_PATH=$LD_LIBRARY_PATH:/usr/local/cuda/lib64
// Next, compile using nvcc:
// $> nvcc hello.c
*/

#define ARRAY_SIZE 10

__global__ void cuda_hello()
  { printf("Hello world!\n"); }

__global__ void printThread()
  {  printf("%i ",threadIdx.x); }

__global__ void increment(const int a, int *b)
  {  *b += a;  }

__global__ void incrementArray(int *array, const int b)
  {  int idx = blockIdx.x  * blockDim.x + threadIdx.x;
     array[idx] += b;  }

int main() {
   // This file needs to be named *.cu or else nvcc compiler will not
   // recognize function<<<x,x>>>(), because it is not standard C/C++ syntax
    cuda_hello<<<1,1>>>();

   // The previous function will not give an output unless/until there
   // is additional computation on the GPU.

  int a=2, *dev_a;   // 'a' will be host data, dev_a will be device data
    // create device/GPU data for dev_a
  hipMalloc((void**)&dev_a, sizeof(int));
    // copy data from host/CPU to device/GPU
  hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyDeviceToHost);
    // load and execute GPU kernel
  increment<<<1,1>>>(5, dev_a);
    // copy data back to CPU
  hipMemcpy(&a, dev_a, sizeof(int), hipMemcpyDeviceToHost);
  std::cout << a << std::endl;    // output should show a=7
  hipFree(dev_a);

  int array[ARRAY_SIZE] = {0};
  int *dev_array;
  hipMalloc((void**)&dev_array, sizeof(int)*ARRAY_SIZE);
  hipMemcpy(dev_array, array, sizeof(int)*ARRAY_SIZE, hipMemcpyHostToDevice);
  incrementArray<<<1,ARRAY_SIZE>>>(dev_array,1);
  hipMemcpy(array, dev_array, sizeof(int)*ARRAY_SIZE, hipMemcpyDeviceToHost);
  for(int i=0; i<ARRAY_SIZE; i++)
    std::cout << array[i] << " ";    // all array elements should be 1
  std::cout << std::endl;
  increment<<<1,1>>>(-1, &dev_array[ARRAY_SIZE-1]);
  hipMemcpy(array, dev_array, sizeof(int)*ARRAY_SIZE, hipMemcpyDeviceToHost);
  for(int i=0; i<ARRAY_SIZE; i++)
    std::cout << array[i] << " ";    // last element should be 0
  std::cout << std::endl;


   // Cuda threads are dentified with threadIdx.x, threadIdx.y, threadIdx.z.
   // This function is 1D, so only uses threadIdx.x
  printThread<<<1,5>>>();

   // The printThread kernel does not require waiting for computation, so we
   //  must wait until it is complete to avoid sync issues
  hipDeviceSynchronize();
  std::cout << std::endl;

  printThread<<<3,5>>>();
  hipDeviceSynchronize();
  std::cout << std::endl;

  hipFree(dev_array);

  return 0;
}
